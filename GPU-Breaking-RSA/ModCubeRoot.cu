
#include <hip/hip_runtime.h>
//The max possible number of roots
#define DEV_MAX_ROOTS_NUM 3

//Number of threads per block.
#define NT 1024

//The array index in global memory.
__device__ int devArrayIndex;

//The answer array stored roots in global memory
__device__ unsigned long long int devAnswer[DEV_MAX_ROOTS_NUM];

/**
 * Device kernel to find (a) cube root(s) for a given integer c with a modular n.
 * <P>
 * Called with a one-dimensional grid of one-dimensional blocks.
 *
 * @param  c        The mod cube (input).
 * @param  n        The module (input).
 *
 * @author  Junan Zhao
 * @version 26-Nov-2018
 */
extern "C" __global__ void modCubeRoot( int c, int n)
{
   //Determine number of threads and this thread's m (test number).
   unsigned long long m = blockIdx.x*NT + threadIdx.x;
   unsigned long long size = gridDim.x*NT;    
   for(; m<n; m+=size)   //use loop to cover all range of n if n is a pretty large integer cannot covered by one round
     {
        unsigned long long temp = m*m;
        temp = temp%n;
        temp = temp*m;
        temp = temp%n;
        if(c==(int)temp) //once found a root
          {             
             int oldIndex = atomicAdd(&devArrayIndex,1);
             devAnswer[oldIndex] = m; 
          }
     }
}